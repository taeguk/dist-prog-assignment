#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <assert.h>
#include <stdio.h>

#include "cuda.cuh"
#include "timer.h"

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;

#define USE_GPU_TIMER 1

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
    CUDA_CALL(hipEventCreate(&cuda_timer_start));
    CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
    CUDA_CALL( hipEventDestroy( cuda_timer_start ) );
    CUDA_CALL( hipEventDestroy( cuda_timer_stop ) );
}

inline void start_device_timer()
{
    hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
    TIMER_T ms;
    hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
    hipEventSynchronize(cuda_timer_stop);

    hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
    return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

__host__ void cuda_error_check(const char * prefix, const char * postfix)
{
    if (hipPeekAtLastError() != hipSuccess)
    {
        printf("%s%s%s", prefix, hipGetErrorString(hipGetLastError()), postfix);
        hipDeviceReset();
        //wait_exit();
        exit(1);
    }
}

static __global__ void cuda_1(float C[MATRIX_SIZE][MATRIX_SIZE], float A[MATRIX_SIZE][MATRIX_SIZE], float B[MATRIX_SIZE][MATRIX_SIZE]);
static __global__ void cuda_2(float C[MATRIX_SIZE][MATRIX_SIZE], float A[MATRIX_SIZE][MATRIX_SIZE], float B[MATRIX_SIZE][MATRIX_SIZE]);
static __global__ void cuda_3(float C[MATRIX_SIZE][MATRIX_SIZE], float A[MATRIX_SIZE][MATRIX_SIZE], float B[MATRIX_SIZE][MATRIX_SIZE]);

extern "C" void cuda_init()
{
    CHECK_TIME_INIT_GPU();
}

extern "C" void cuda_destroy()
{
    CHECK_TIME_DEST_GPU();
}

extern "C" void cuda_version_1(float C[MATRIX_SIZE][MATRIX_SIZE], float A[MATRIX_SIZE][MATRIX_SIZE], float B[MATRIX_SIZE][MATRIX_SIZE])
{
    void *A_device, *B_device, *C_device;
    double start, finish;

    GET_TIME(start);

    CUDA_CALL(hipMalloc((void**)&A_device, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE));
    CUDA_CALL(hipMalloc((void**)&B_device, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE));
    CUDA_CALL(hipMalloc((void**)&C_device, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE));

    // transfer data from host to device.
    CUDA_CALL(hipMemcpy(A_device, A, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(B_device, B, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyHostToDevice));

    // execute kernel.
    dim3 block(32,32);
    dim3 grid(MATRIX_SIZE/32, MATRIX_SIZE/32);
    float device_time;

    printf("[CUDA 1] Start Launching Kernel (cuda_1).\n");
    CHECK_TIME_START_GPU();
    cuda_1<<<grid, block>>>((float(*)[MATRIX_SIZE])C_device, (float(*)[MATRIX_SIZE])A_device, (float(*)[MATRIX_SIZE])B_device);
    cuda_error_check( "ERROR: ", " when cuda_1 was launched.\n" );
    CHECK_TIME_END_GPU(device_time);
    printf("[CUDA 1] Only Processing Elapsed Time : %.3f (sec).\n", device_time/1000);

    // transfer result from device to host.
    CUDA_CALL(hipMemcpy(C, C_device, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyDeviceToHost));
    CUDA_CALL( hipDeviceSynchronize() );

    GET_TIME(finish);

    printf("[CUDA 1] All Elapsed Time : %.3f (sec).\n", finish-start);
}

extern "C" void cuda_version_2(float C[MATRIX_SIZE][MATRIX_SIZE], float A[MATRIX_SIZE][MATRIX_SIZE], float B[MATRIX_SIZE][MATRIX_SIZE])
{
    void *A_device, *B_device, *C_device;
    double start, finish;

    GET_TIME(start);

    CUDA_CALL(hipMalloc((void**)&A_device, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE));
    CUDA_CALL(hipMalloc((void**)&B_device, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE));
    CUDA_CALL(hipMalloc((void**)&C_device, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE));

    // transfer data from host to device.
    CUDA_CALL(hipMemcpy(A_device, A, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(B_device, B, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyHostToDevice));

    // execute kernel.
    dim3 block(32,32);
    dim3 grid(MATRIX_SIZE/32, MATRIX_SIZE/32);
    float device_time;

    printf("[CUDA 2] Start Launching Kernel (cuda_2).\n");
    CHECK_TIME_START_GPU();
    cuda_2<<<grid, block, sizeof(float)*32*32>>>((float(*)[MATRIX_SIZE])C_device, (float(*)[MATRIX_SIZE])A_device, (float(*)[MATRIX_SIZE])B_device);
    cuda_error_check( "ERROR: ", " when cuda_2 was launched.\n" );
    CHECK_TIME_END_GPU(device_time);
    printf("[CUDA 2] Only Processing Elapsed Time : %.3f (sec).\n", device_time/1000);

    // transfer result from device to host.
    CUDA_CALL(hipMemcpy(C, C_device, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyDeviceToHost));
    CUDA_CALL( hipDeviceSynchronize() );

    GET_TIME(finish);

    printf("[CUDA 2] All Elapsed Time : %.3f (sec).\n", finish-start);
}

extern "C" void cuda_version_3(float C[MATRIX_SIZE][MATRIX_SIZE], float A[MATRIX_SIZE][MATRIX_SIZE], float B[MATRIX_SIZE][MATRIX_SIZE])
{
    void *A_device, *B_device, *C_device;
    double start, finish;

    GET_TIME(start);

    CUDA_CALL(hipMalloc((void**)&A_device, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE));
    CUDA_CALL(hipMalloc((void**)&B_device, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE));
    CUDA_CALL(hipMalloc((void**)&C_device, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE));

    // transfer data from host to device.
    CUDA_CALL(hipMemcpy(A_device, A, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(B_device, B, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyHostToDevice));

    // execute kernel.
    dim3 block(32,32);
    dim3 grid(MATRIX_SIZE/32, MATRIX_SIZE/32);
    float device_time;

    printf("[CUDA 3] Start Launching Kernel (cuda_3).\n");
    CHECK_TIME_START_GPU();
    cuda_3<<<grid, block, 2*sizeof(float)*32*32>>>((float(*)[MATRIX_SIZE])C_device, (float(*)[MATRIX_SIZE])A_device, (float(*)[MATRIX_SIZE])B_device);
    cuda_error_check( "ERROR: ", " when cuda_3 was launched.\n" );
    CHECK_TIME_END_GPU(device_time);
    printf("[CUDA 3] Only Processing Elapsed Time : %.3f (sec).\n", device_time/1000);

    // transfer result from device to host.
    CUDA_CALL(hipMemcpy(C, C_device, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyDeviceToHost));
    CUDA_CALL( hipDeviceSynchronize() );

    GET_TIME(finish);

    printf("[CUDA 3] All Elapsed Time : %.3f (sec).\n", finish-start);
}

static __global__ void cuda_1(float C[MATRIX_SIZE][MATRIX_SIZE], float A[MATRIX_SIZE][MATRIX_SIZE], float B[MATRIX_SIZE][MATRIX_SIZE])
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float result = 0.0f;
    for(int k=0; k<MATRIX_SIZE; ++k)
        result += A[row][k] * B[k][col];
    C[row][col] = result;
}

static __global__ void cuda_2(float C[MATRIX_SIZE][MATRIX_SIZE], float A[MATRIX_SIZE][MATRIX_SIZE], float B[MATRIX_SIZE][MATRIX_SIZE])
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ float B_shared[/* blockDim.y * blockDim.x */];

    float result = 0.0f;
    int y_base = 0;
    while (y_base < MATRIX_SIZE) {
        // load a chunk from global memory to shared memory.
        B_shared[threadIdx.y * blockDim.x + threadIdx.x] = B[y_base + threadIdx.y][col];
        //B_shared[threadIdx.x * blockDim.y + threadIdx.y] = B[y_base + threadIdx.y][col];
        //B_shared[threadIdx.x * (blockDim.y+1) + threadIdx.y] = B[y_base + threadIdx.y][col];
        __syncthreads();

        for(int k=0; k<blockDim.y; ++k)
            //result += A[row][y_base + k] * B[y_base + k][col];
            result += A[row][y_base + k] * B_shared[k * blockDim.x + threadIdx.x];    // most fastest.
        //result += A[row][y_base + k] * B_shared[threadIdx.x * blockDim.y + k];  // bank conflict.
        //result += A[row][y_base + k] * B_shared[threadIdx.x * (blockDim.y+1) + k];  // no bank conflict.

        // * THIS BARRIER MUST BE NEEDED! because of B_shared.
        __syncthreads();

        y_base += blockDim.y;
    }
    C[row][col] = result;
}

static __global__ void cuda_3(float C[MATRIX_SIZE][MATRIX_SIZE], float A[MATRIX_SIZE][MATRIX_SIZE], float B[MATRIX_SIZE][MATRIX_SIZE])
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ float shared_memory[/* 2 * blockDim.y * blockDim.x */];
    float *A_shared = &shared_memory[0];
    float *B_shared = &shared_memory[blockDim.y * blockDim.x];

    float result = 0.0f;
    int k_base = 0;
    while (k_base < MATRIX_SIZE) {
        // load a chunk from global memory to shared memory.
        A_shared[threadIdx.y * blockDim.x + threadIdx.x] = A[row][k_base + threadIdx.x];
        B_shared[threadIdx.y * blockDim.x + threadIdx.x] = B[k_base + threadIdx.y][col];
        __syncthreads();

        for(int k=0; k<blockDim.y; ++k)
            result += A_shared[threadIdx.y * blockDim.x + k] * B_shared[k * blockDim.x + threadIdx.x];

        __syncthreads();

        k_base += blockDim.y;
    }
    C[row][col] = result;
}
