#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <assert.h>
#include <stdio.h>

#include "cuda.cuh"
#include "timer.h"

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;

#define USE_GPU_TIMER 1

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
    CUDA_CALL(hipEventCreate(&cuda_timer_start));
    CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
    CUDA_CALL( hipEventDestroy( cuda_timer_start ) );
    CUDA_CALL( hipEventDestroy( cuda_timer_stop ) );
}

inline void start_device_timer()
{
    hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
    TIMER_T ms;
    hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
    hipEventSynchronize(cuda_timer_stop);

    hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
    return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

__host__ void cuda_error_check(const char * prefix, const char * postfix)
{
    if (hipPeekAtLastError() != hipSuccess)
    {
        printf("%s%s%s", prefix, hipGetErrorString(hipGetLastError()), postfix);
        hipDeviceReset();
        //wait_exit();
        exit(1);
    }
}

static __global__ void cuda_1(int *arr, int *output, int size);
static __global__ void cuda_2(int *arr, int *output, int size);
static __global__ void cuda_3(int *arr, int *output, int size);
static __global__ void cuda_4(int *arr, int *output, int size);

extern "C" void cuda_init()
{
    CHECK_TIME_INIT_GPU();
}

extern "C" void cuda_destroy()
{
    CHECK_TIME_DEST_GPU();
}

extern "C" void cuda_version_1(int *res, int arr[N])
{
    void *output_device, *arr_device;
    double start, finish;
    const int num_threads = 256;
    int size = N;
    int num_blocks = (size+num_threads-1) / num_threads;

    GET_TIME(start);

    CUDA_CALL(hipMalloc((void**)&arr_device, sizeof(int) * N));
    CUDA_CALL(hipMalloc((void**)&output_device, sizeof(int) * num_blocks));

    // transfer data from host to device.
    CUDA_CALL(hipMemcpy(arr_device, arr, sizeof(int) * N, hipMemcpyHostToDevice));

    float all_device_time = 0.f;

    do {
        float device_time;

        CHECK_TIME_START_GPU();
        cuda_1<<<num_blocks, num_threads>>>((int*)arr_device, (int*)output_device, size);
        cuda_error_check( "ERROR: ", " when cuda_1 was launched.\n" );
        CHECK_TIME_END_GPU(device_time);
        all_device_time += device_time;

        {
            void *tmp = arr_device;
            arr_device = output_device;
            output_device = tmp;
        }
        size = num_blocks;
        num_blocks = (size+num_threads-1) / num_threads;

    } while (size > 1);

    printf("[CUDA 1] Only Processing Elapsed Time : %.3f (sec).\n", all_device_time/1000);

    // transfer result from device to host.
    CUDA_CALL(hipMemcpy(res, arr_device, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CALL( hipDeviceSynchronize() );

    CUDA_CALL(hipFree(arr_device));
    CUDA_CALL(hipFree(output_device));

    GET_TIME(finish);

    printf("[CUDA 1] All Elapsed Time : %.3f (sec).\n", finish-start);
}

extern "C" void cuda_version_2(int *res, int arr[N])
{
    void *output_device, *arr_device;
    double start, finish;
    const int num_threads = 256;
    int size = N;
    int num_blocks = (size+num_threads-1) / num_threads;

    GET_TIME(start);

    CUDA_CALL(hipMalloc((void**)&arr_device, sizeof(int) * N));
    CUDA_CALL(hipMalloc((void**)&output_device, sizeof(int) * num_blocks));

    // transfer data from host to device.
    CUDA_CALL(hipMemcpy(arr_device, arr, sizeof(int) * N, hipMemcpyHostToDevice));

    float all_device_time = 0.f;

    do {
        float device_time;

        CHECK_TIME_START_GPU();
        cuda_2<<<num_blocks, num_threads>>>((int*)arr_device, (int*)output_device, size);
        cuda_error_check( "ERROR: ", " when cuda_2 was launched.\n" );
        CHECK_TIME_END_GPU(device_time);
        all_device_time += device_time;

        {
            void *tmp = arr_device;
            arr_device = output_device;
            output_device = tmp;
        }
        size = num_blocks;
        num_blocks = (size+num_threads-1) / num_threads;

    } while (size > 1);

    printf("[CUDA 2] Only Processing Elapsed Time : %.3f (sec).\n", all_device_time/1000);

    // transfer result from device to host.
    CUDA_CALL(hipMemcpy(res, arr_device, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CALL( hipDeviceSynchronize() );

    CUDA_CALL(hipFree(arr_device));
    CUDA_CALL(hipFree(output_device));

    GET_TIME(finish);

    printf("[CUDA 2] All Elapsed Time : %.3f (sec).\n", finish-start);
}

extern "C" void cuda_version_3(int *res, int arr[N])
{
    void *output_device, *arr_device;
    double start, finish;
    const int num_threads = 256;
    int size = N;
    int num_blocks = (size+num_threads-1) / num_threads;

    GET_TIME(start);

    CUDA_CALL(hipMalloc((void**)&arr_device, sizeof(int) * N));
    CUDA_CALL(hipMalloc((void**)&output_device, sizeof(int) * num_blocks));

    // transfer data from host to device.
    CUDA_CALL(hipMemcpy(arr_device, arr, sizeof(int) * N, hipMemcpyHostToDevice));

    float all_device_time = 0.f;

    do {
        float device_time;

        CHECK_TIME_START_GPU();
        cuda_3<<<num_blocks, num_threads, sizeof(int) * num_threads>>>((int*)arr_device, (int*)output_device, size);
        cuda_error_check( "ERROR: ", " when cuda_3 was launched.\n" );
        CHECK_TIME_END_GPU(device_time);
        all_device_time += device_time;

        {
            void *tmp = arr_device;
            arr_device = output_device;
            output_device = tmp;
        }
        size = num_blocks;
        num_blocks = (size+num_threads-1) / num_threads;

    } while (size > 1);

    printf("[CUDA 3] Only Processing Elapsed Time : %.3f (sec).\n", all_device_time/1000);

    // transfer result from device to host.
    CUDA_CALL(hipMemcpy(res, arr_device, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CALL( hipDeviceSynchronize() );

    CUDA_CALL(hipFree(arr_device));
    CUDA_CALL(hipFree(output_device));

    GET_TIME(finish);

    printf("[CUDA 3] All Elapsed Time : %.3f (sec).\n", finish-start);
}

extern "C" void cuda_version_4(int *res, int arr[N])
{
    void *output_device, *arr_device;
    double start, finish;
    const int num_threads = 256;
    int size = N;
    int num_blocks = (size+num_threads-1) / num_threads;

    GET_TIME(start);

    CUDA_CALL(hipMalloc((void**)&arr_device, sizeof(int) * N));
    CUDA_CALL(hipMalloc((void**)&output_device, sizeof(int) * num_blocks));

    // transfer data from host to device.
    CUDA_CALL(hipMemcpy(arr_device, arr, sizeof(int) * N, hipMemcpyHostToDevice));

    float all_device_time = 0.f;

    do {
        float device_time;

        CHECK_TIME_START_GPU();
        cuda_4<<<num_blocks, num_threads, sizeof(int) * num_threads>>>((int*)arr_device, (int*)output_device, size);
        cuda_error_check( "ERROR: ", " when cuda_4 was launched.\n" );
        CHECK_TIME_END_GPU(device_time);
        all_device_time += device_time;

        {
            void *tmp = arr_device;
            arr_device = output_device;
            output_device = tmp;
        }
        size = num_blocks;
        num_blocks = (size+num_threads-1) / num_threads;

    } while (size > 1);

    printf("[CUDA 4] Only Processing Elapsed Time : %.3f (sec).\n", all_device_time/1000);

    // transfer result from device to host.
    CUDA_CALL(hipMemcpy(res, arr_device, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CALL( hipDeviceSynchronize() );

    CUDA_CALL(hipFree(arr_device));
    CUDA_CALL(hipFree(output_device));

    GET_TIME(finish);

    printf("[CUDA 4] All Elapsed Time : %.3f (sec).\n", finish-start);
}

static __global__ void cuda_1(int *arr, int *output, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for(int s=1; s<blockDim.x; s*=2) {
        if (threadIdx.x % (s*2) == 0 && tid+s < size)     // path divergence
            arr[tid] = max(arr[tid], arr[tid+s]);
        __syncthreads();
    }
    if(threadIdx.x == 0)
        output[blockIdx.x] = arr[tid];
}

static __global__ void cuda_2(int *arr, int *output, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int base = blockIdx.x * blockDim.x;

    for(int s=1; s<blockDim.x; s*=2) {
        int offset = 2*s*threadIdx.x;
        int idx = base + offset;
        if (offset < blockDim.x  && idx + s < size)    // no path divergence
            arr[idx] = max(arr[idx], arr[idx + s]);
        __syncthreads();
    }
    if(threadIdx.x == 0)
        output[blockIdx.x] = arr[tid];
}

static __global__ void cuda_3(int *arr, int *output, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ int arr_shared[/* blockDim.x */];
    int shared_size = min(blockDim.x, size - blockIdx.x * blockDim.x);    // size of arr_shared.

    arr_shared[threadIdx.x] = arr[tid];
    __syncthreads();

    for(int s=1; s<blockDim.x; s*=2) {
        int idx = 2*s*threadIdx.x;
        if (idx < blockDim.x && idx + s < shared_size)
            arr_shared[idx] = max(arr_shared[idx], arr_shared[idx + s]);    // bank conflict
        __syncthreads();
    }
    if(threadIdx.x == 0)
        output[blockIdx.x] = arr_shared[0];
}

static __global__ void cuda_4(int *arr, int *output, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ int arr_shared[/* blockDim.x */];
    int shared_size = min(blockDim.x, size - blockIdx.x * blockDim.x);    // size of arr_shared.

    arr_shared[threadIdx.x] = arr[tid];
    __syncthreads();

    for(int s=blockDim.x/2; s>0; s/=2) {
        if (threadIdx.x < s && threadIdx.x + s < shared_size)
            arr_shared[threadIdx.x] = max(arr_shared[threadIdx.x], arr_shared[threadIdx.x + s]);    // no bank conflict
        __syncthreads();
    }
    if(threadIdx.x == 0)
        output[blockIdx.x] = arr_shared[0];
}
